/************************************************************************************
 *   This file is part of cudatest.                                                 *
 *                                                                                  *
 *   MIT License                                                                    *
 *                                                                                  *
 *   Copyright (c) 2018 Ivo Filot <i.a.w.filot@tue.nl>                              *
 *                                                                                  *
 *   Permission is hereby granted, free of charge, to any person obtaining a copy   *
 *   of this software and associated documentation files (the "Software"), to deal  *
 *   in the Software without restriction, including without limitation the rights   *
 *   to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      *
 *   copies of the Software, and to permit persons to whom the Software is          *
 *   furnished to do so, subject to the following conditions:                       *
 *                                                                                  *
 *   The above copyright notice and this permission notice shall be included in all *
 *   copies or substantial portions of the Software.                                *
 *                                                                                  *
 *   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     *
 *   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       *
 *   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    *
 *   AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         *
 *   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  *
 *   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  *
 *   SOFTWARE.                                                                      *
 *                                                                                  *
 ************************************************************************************/

#include "card_manager.h"

CardManager::CardManager() {}

void CardManager::probe_cards() {
    int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device Number: " << i << std::endl;
		std::cout << "  Device name: " << prop.name << std::endl;
		std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
		std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
		std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl << std::endl;
	}
}
